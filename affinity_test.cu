﻿
// CUDA driver & runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// C++
#include <cstdint>
#include <iostream>
#include <numeric>
#include <vector>

#define cuda_try(call)                                                                \
  do {                                                                                \
    hipError_t err = static_cast<hipError_t>(call);                                 \
    if (err != hipSuccess) {                                                         \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorName(err)); \
      std::terminate();                                                               \
    }                                                                                 \
  } while (0)

hipDevice_t get_cuda_device(const int device_id, int& sms_count) {
  hipDevice_t device;
  int device_count = 0;

  cuda_try(hipInit(0));  // Flag parameter must be zero
  cuda_try(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    std::cout << "No CUDA capable device found." << std::endl;
    std::terminate();
  }

  cuda_try(hipDeviceGet(&device, device_id));

  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, device_id);

  sms_count = device_prop.multiProcessorCount;

  std::cout << "Device[" << device_id << "]: " << device_prop.name << '\n';
  std::cout << "SMs count: " << sms_count << '\n';

  return device;
}

int main(int argc, char** argv) {
  int device_id = 0;
  int sms_count = 0;
  hipDevice_t dev = get_cuda_device(device_id, sms_count);

  float load = 0.5f;
  int load_sms = static_cast<int>(load * sms_count);
  hipCtx_t ctx;
  CUexecAffinityParam_v1 affinity_param{
      CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, load_sms};
  auto flags = CUctx_flags::hipDeviceScheduleAuto;

  cuda_try(cuCtxCreate_v3(&ctx, &affinity_param, 1, flags, dev));

  cuda_try(hipCtxDestroy(ctx));
}
